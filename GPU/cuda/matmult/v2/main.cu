#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

void printMatrixCustom(double* matrix,size_t size_r,size_t size_c){
    printf("\n *************** MATRIX ****************\n\n");
    int i,j;
    for(i = 0; i < size_r; i++) {
        for (j = 0; j < size_c; ++j) {
            printf(" %f ",matrix[i*size_c+j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 1000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./matmult               # Matrix of size 1,000 x 1,000 is used"
           "\n    Usage: ./matmult <m>           # Matrix of size m x m is used"
           "\n");
        exit(0);
    }

    double* A_h = (double*) malloc( sizeof(double)*n*n );
    for (unsigned int i=0; i < n*n; i++) { A_h[i] = (rand()%100)/100.00; }

    double* B_h = (double*) malloc( sizeof(double)*n*n );
    for (unsigned int i=0; i < n*n; i++) { B_h[i] = (rand()%100)/100.00; }

    double* C_h = (double*) malloc( sizeof(double)*n*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Matrix size = %u x %u\n", n, n);
    
    //printMatrixCustom(A_h,n,n);
    //printMatrixCustom(B_h,n,n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	int size = n*n*sizeof(double);

	double* A_d;
	hipMalloc((void**)&A_d,size);

	double* B_d;
	hipMalloc((void**)&B_d,size);

	double* C_d;
	hipMalloc((void**)&C_d,size);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	int BLOCK_SIZE = 32;

	// llamada al kernel
	dim3 thrds(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((int)ceil((double)n/thrds.x), (int)ceil((double)n/thrds.y));
	
	matmultKernel<<<grid,thrds>>>(A_d, B_d, C_d,n);
	//printf("\n Error:\t %s\n", hipGetErrorString(hipGetLastError()));

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    //printMatrixCustom(C_h,n,n);

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);


    return 0;

}

