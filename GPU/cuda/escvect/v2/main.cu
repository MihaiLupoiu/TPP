#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./escvect               # Vector of size 10,000 is used"
           "\n    Usage: ./escvect <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( 1*sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float result_h = 0;

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	int size = n*sizeof(float);

	float* A_d;
	hipMalloc((void**)&A_d,size);

	float* B_d;
	hipMalloc((void**)&B_d,size);

	float* result_d;
	hipMalloc((void**)&result_d,sizeof(float));

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	int nhilos = 256;
	int nbloques = ceil(float(n)/nhilos);

	escvectKernel<<<nbloques,nhilos>>>(A_d, B_d, result_d, n);


    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

	hipMemcpy(&result_h, result_d, sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

	//printf("\nResult: %f\n",result_h);
	
    verify(A_h, B_h, result_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);

    //INSERT CODE HERE

	hipFree(A_d);
	hipFree(B_d);
	hipFree(result_d);


    return 0;

}

