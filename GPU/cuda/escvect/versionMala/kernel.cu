
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ void escvectKernel(float* A, float* B, float* result, int n) {

	float res = 0;

	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i<n){
		res += A[i]*B[i];
		//printf("A[%d] = %f, B[%d] = %f, res = %f\n",i,A[i],i,B[i],res);
	}

	*result =  *result + res;
}

